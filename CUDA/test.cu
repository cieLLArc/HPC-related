
#include <hip/hip_runtime.h>
#include <stdio.h>

#define size 10

void compute_cpu(int *A, int *B, int *C)
{
    for (int i = 0; i < size; i++)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void compute_gpu(int *gpuA, int *gpuB, int *gpuC)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    gpuC[tid] = gpuA[tid] + gpuB[tid];
}

void print_list(int *C)
{
    for (int i = 0; i < size; i++)
    {
        printf("%d ", C[i]);
    }
    printf("\n");
}

int main()
{
    int *A = (int *)malloc(size * sizeof(int));
    int *B = (int *)malloc(size * sizeof(int));
    int *C = (int *)malloc(size * sizeof(int));

    if (A && B && C)
    {
        memset(A, 0, size * sizeof(int));
        memset(B, 0, size * sizeof(int));
        memset(C, 0, size * sizeof(int));
    }
    else
    {
        free(A);
        free(B);
        free(C);
        printf("MEMORY failed to allocate\n");
        exit(-1);
    }
    for (int i = 0; i < size; i++)
    {
        A[i] = i;
        B[i] = i;
    }

    printf("computed on cpu\n");
    compute_cpu(A, B, C);
    print_list(C);

    // 在gpu global memory中开辟数组空间

    int *gpuA, *gpuB, *gpuC;
    hipMalloc((int **)&gpuA, size * sizeof(int));
    hipMalloc((int **)&gpuB, size * sizeof(int));
    hipMalloc((int **)&gpuC, size * sizeof(int));

    if (gpuA && gpuB && gpuC)
    {
        hipMemset(gpuA, 0, size * sizeof(int));
        hipMemset(gpuB, 0, size * sizeof(int));
        hipMemset(gpuC, 0, size * sizeof(int));
    }
    else
    {
        hipFree(gpuA);
        hipFree(gpuB);
        hipFree(gpuC);
        printf("GMEMORY failed to allocate\n");
        exit(-1);
    }

    // 将数据从主存cpu(host)传输到显存gpu(device)
    hipMemcpy(gpuA, A, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuB, B, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuC, C, size * sizeof(int), hipMemcpyHostToDevice);

    compute_gpu<<<1, size>>>(gpuA, gpuB, gpuC);

    hipDeviceSynchronize();

    // 将数据从device传回host
    int *D = (int *)malloc(size * sizeof(int));
    hipMemcpy(D, gpuC, size * sizeof(int), hipMemcpyDeviceToHost);
    printf("computed on gpu\n");
    print_list(D);

    free(A);
    free(B);
    free(C);
    free(D);
    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuC);

    return 0;
}