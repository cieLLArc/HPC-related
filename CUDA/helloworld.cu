
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
    printf("%d %d\n", blockIdx.x, threadIdx.x);
}
int main()
{
    // Launch kernel
    helloFromGPU<<<3, 2>>>();
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    printf("Hello World from CPU!\n");
    return 0;
}
